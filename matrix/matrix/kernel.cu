#include "hip/hip_runtime.h"
// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include ""
// CUDA and CUBLAS functions
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_string.h>
#include <stdio.h>
#include "setMatrix.h"
#include "printD.h"
#include "MatrixFunction.cuh"


int main(int argc, char *argv[])
{
	int DIMENSIONS = 2;

	int num_samplingPoints; // �����������
	int width_originalPoints; // ԭʼ������ĺ����������
	int height_originalPoints; // ԭʼ������������������
	int num_originalPoints; //
	float *m_samplingPoints; // �������λ�ã�
	float *m_originalPoints; // ԭʼ�ܶȺ�������ɢ�㣻
	float *m_samplingPointsDensity; // ��������ܶȣ�
	float *m_originalPointsDesntiy;

	num_samplingPoints = setNumSamplingPoint();
	width_originalPoints = setWidthOriginalPoint();
	height_originalPoints = setHeightOriginalPoint();
	num_originalPoints = width_originalPoints * height_originalPoints;

	m_samplingPoints = (float *)malloc(num_samplingPoints * DIMENSIONS * sizeof(float));
	m_originalPoints = (float *)malloc(width_originalPoints * height_originalPoints * DIMENSIONS * sizeof(float));
	m_samplingPointsDensity = (float *)malloc(num_samplingPoints * sizeof(float));
	m_originalPointsDesntiy = (float *)malloc(num_originalPoints * sizeof(float));

	setSamplingPoints(m_samplingPoints, num_samplingPoints, DIMENSIONS);
	setOriginalPoints(m_originalPoints, height_originalPoints, width_originalPoints, DIMENSIONS);
	setSamplingPointDensity(m_samplingPointsDensity, num_samplingPoints);
	setSamplingPointDensity(m_originalPointsDesntiy, num_originalPoints);
	
	//�������
	write2File(m_originalPoints, width_originalPoints * height_originalPoints * DIMENSIONS,1);
	write2File(m_samplingPoints, num_samplingPoints* DIMENSIONS, 2);
	write2File(m_originalPointsDesntiy, num_originalPoints, 3);
	write2File(m_samplingPointsDensity, num_samplingPoints, 4);

	int block_size;

	// ��ʼ��CUDA
	block_size = initCuda();

	dim3 dims_sampling_points(num_samplingPoints, 2, 1);
	dim3 dims_original_points(width_originalPoints * height_originalPoints, 2, 1);
	// Add vectors in parallel.

	int _result = distanceCompuation(block_size, dims_sampling_points, dims_original_points, m_samplingPoints, m_originalPoints, m_samplingPointsDensity, m_originalPointsDesntiy);

	free(m_samplingPoints);
	free(m_originalPoints);
	free(m_originalPointsDesntiy);
	free(m_samplingPointsDensity);
	exit(_result);


}

int distanceCompuation(int block_size, dim3 &dimsA, dim3 &dimsB, float *matrix_A, float *matrix_B, float *matrix_C, float *matrix_D){
	// allocate host memory for original points and sampling points

	float *theta = (float *)malloc(sizeof(float));
	*theta = 0.5;
	float *one_minusTheta = (float *)malloc(sizeof(float));
	*one_minusTheta = 1 - *theta;
	float *lamb = (float *)malloc(sizeof(float));
	*lamb = 1;
	float *_R = (float *)malloc(sizeof(float));
	*_R = 1;

	float *theta_lambR = (float *)malloc(sizeof(float));
	float *minusOne_divLabR = (float *)malloc(sizeof(float));

	*theta_lambR = *theta * *lamb * *_R;
	*minusOne_divLabR = -1 / *lamb / *_R;


	float stop_U = 0.001; // ���㴫��ƻ������ֹͣ��׼
	float stop_X = 0.001;// ���������ֹͣ��׼

	float alpha = 1.0;
	float beta = 0.0;
	unsigned int _iter = 20;
	float temp_alpha = -1.0;

	unsigned int size_A = dimsA.x * dimsA.y * dimsA.z;
	unsigned int mem_sizeA = sizeof(float) * size_A;
	float *h_A = matrix_A; // �����������

	unsigned int size_B = dimsB.x * dimsB.y * dimsB.z;
	unsigned int mem_sizeB = sizeof(float) * size_B;
	float  *h_B = matrix_B; // ������������꣬ԭʼͼ�������

	unsigned int size_samplingPoint = dimsA.x;
	unsigned int mem_sizeSamplingPoint = sizeof(float) * size_samplingPoint;
	float *h_samplingPointDensity = matrix_C;
	
	//h_samplingPointDensity_samplingPoint���ں�������������
	float *h_samplingPointDensity_samplingPoint = (float *)malloc(sizeof(float) * size_samplingPoint);
	for (int i = 0; i < size_samplingPoint; i++){
		h_samplingPointDensity_samplingPoint[i] = 1 / h_samplingPointDensity[i];
	}
	//h_samplingPointDensity_samplingPoint���Ƶ�GPU
	float *d_samplingPointDensity_samplingPoint;
	hipMalloc((void**)&d_samplingPointDensity_samplingPoint, sizeof(float) * size_samplingPoint);
	hipMemcpy(d_samplingPointDensity_samplingPoint, h_samplingPointDensity_samplingPoint, sizeof(float) * size_samplingPoint, hipMemcpyHostToDevice);
	
	unsigned int size_originalPoint = dimsB.x;
	unsigned int mem_sizeOriginalPoint = sizeof(float) * size_originalPoint;
	float *h_originalPointDensity = matrix_D;

	unsigned int size_transportMatrix = dimsA.x * dimsB.x;
	unsigned int mem_sizeTransportMatrix = size_transportMatrix * sizeof(float);

	float *h_V = (float *)malloc(mem_sizeOriginalPoint);// ���㴫��ƻ�������v��
	for (int i = 0; i < dimsB.x; i++){
		*(h_V + i) = (float)1.0;
	}

	float *h_distanceMatrix;
	float *h_kasaiMatrix; // �������ĸ�˹����
	float *h_transportPlan;// = (float *)malloc(dimsA.x * dimsB.x * sizeof(float));

	// allocate device memory 
	float *d_A, *d_B, *d_distanceMatrix, *d_kasaiMatrix, *d_transportPlan, *d_U, *d_V; // device memory�еı���������d_U, d_V Ϊ�м����
	float *d_kasaiV, *d_kasaiU;
	float *d_samplingPointDensity, *d_originalPointDensity;
	float *d_tempVectorStopCri;
	float *d_diagUKasaiMatrix; /// ��ʱ����
	float *d_transportPlanDensity;///��ʱ����
	float *d_tempSamplPointCoordinate; // ��ʱ����
	hipblasHandle_t handle;
	hipblasStatus_t stat;
	hipError_t error;


	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS){
		printf("CUBLAS initialization failed\n");
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_A, mem_sizeA);

	if (error != hipSuccess){
		printf("hipMalloc d_A returned error %s (code %d), line(%d)\n", hipGetErrorString, error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_B, mem_sizeB);

	if (error != hipSuccess){
		printf("hipMalloc d_B returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_distanceMatrix, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_distanceMatrix returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_kasaiMatrix, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_kasaiMatrix returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_U, mem_sizeSamplingPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_U returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_V, mem_sizeOriginalPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_V returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_kasaiV, mem_sizeSamplingPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_kasaiV returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_kasaiU, mem_sizeOriginalPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_kasaiU returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_samplingPointDensity, mem_sizeSamplingPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_samplingPointDensity returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_originalPointDensity, mem_sizeOriginalPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_originalPointDensity returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_diagUKasaiMatrix, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_diagUKasaiMatrix returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_transportPlan, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_transportPlan returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_transportPlanDensity, mem_sizeTransportMatrix);
	if (error != hipSuccess){
		printf("hipMalloc d_transportPlanDensity returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_tempSamplPointCoordinate, mem_sizeA);
	if (error != hipSuccess){
		printf("hipMalloc d_tempSamplPointCoordinate returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMalloc((void**)&d_tempVectorStopCri, mem_sizeSamplingPoint);
	if (error != hipSuccess){
		printf("hipMalloc d_tempVectorStopCri returned error %s(code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// copy host memory to device
	error = hipMemcpy(d_A, h_A, mem_sizeA, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_A, h_A) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_B, h_B, mem_sizeB, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_B, h_B) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_V, h_V, mem_sizeOriginalPoint, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_V, h_V) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_samplingPointDensity, h_samplingPointDensity, mem_sizeSamplingPoint, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_samplingPointDensity, h_samplingPointDensity) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(d_originalPointDensity, h_originalPointDensity, mem_sizeOriginalPoint, hipMemcpyHostToDevice);
	if (error != hipSuccess){
		printf("hipMemcpy (d_originalPointDensity, h_originalPointDensity) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	// ����event
	hipEvent_t start;
	error = hipEventCreate(&start);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	hipEvent_t stop;
	error = hipEventCreate(&stop);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	// Record the start event
	error = hipEventRecord(start, NULL);

	if (error != hipSuccess){
		fprintf(stderr, " Failed to record start evern (error code %s)! \n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float *stop_valueU = (float *)malloc(sizeof(float));
	float *stop_valueX = (float *)malloc(sizeof(float));
	float *transportCost = (float *)malloc(sizeof(float));
	float *preTransportCost = (float *)malloc(sizeof(float));
	*preTransportCost = 99999;
	*stop_valueU = 999999;
	*stop_valueX = 999999;

	//block_size = 32;
	dim3 threads(1, block_size, 1);
	dim3 grid(dimsA.x, dimsB.x / block_size, 1);

	// ����������
	if (block_size == 16){
		distancePointToPointCUDA<16> << <grid, threads >> >(d_distanceMatrix, d_A, d_B, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
	}
	else{
		distancePointToPointCUDA<32> << <grid, threads >> >(d_distanceMatrix, d_A, d_B, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
	}
	hipDeviceSynchronize();// ͬ������
	
/*
	// ����distancePointToPointCUDA�Ƿ���ȷ 
	
	h_distanceMatrix = (float *)malloc(mem_sizeTransportMatrix);
	error = hipMemcpy(h_distanceMatrix, d_distanceMatrix, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_distanceMatrix, d_distanceMatrix) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}
	// CPU ����������
	float *c_C = (float *)malloc(mem_sizeTransportMatrix);
	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++){
			float diff_x;
			float diff_y;
			diff_x = *(h_A + i * dimsA.y) - *(h_B + j*dimsB.y);
			diff_y = *(h_A + i * dimsA.y + 1) - *(h_B + j*dimsB.y + 1);
			*(c_C + dimsB.x * i + j) = diff_x * diff_x + diff_y * diff_y;
		}
	}
	printf("The distance matrix: GPU  CPU.\n");
	for (int i = 0; i < dimsA.x * dimsB.x; i++){
		printf("The distance matrix: %f  %f \n", h_distanceMatrix[i], c_C[i]);
	}

	float diff_ = 0;
	for (int i = 0; i < dimsA.x; i++){
		for (int j = 0; j < dimsB.x; j++)
			diff_ += abs(h_distanceMatrix[i * dimsB.x + j] - c_C[i*dimsB.x + j]);
	}
	printf("The difference distance computation between results of CPU and GPU is %f.\n", diff_);
	//	free(c_C);
*/	

	float *h_kasaiV = (float *)malloc(mem_sizeSamplingPoint);
	float *h_kasaiU = (float *)malloc(mem_sizeOriginalPoint);
	//float *h_V = (float *)malloc(mem_sizeOriginalPoint);
	float *h_U = (float *)malloc(mem_sizeSamplingPoint);

	//ͳ��ʱ��
	//hipEvent_t start, stop;
	float time;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//while (*stop_valueX > 0.0){
	for (int ii = 0; ii < 20;ii++){
		// ����Kasai����
		threads.x = block_size;
		threads.y = 1;
		threads.z = 1;
		grid.x = ceil(float(size_transportMatrix) / threads.x);
		grid.y = 1;
		grid.z = 1;

		if (block_size == 16){
			kaisaiMatrixComputation<4> << <grid, threads >> >(d_kasaiMatrix, d_distanceMatrix);
		}
		else{
			kaisaiMatrixComputation<32> << <grid, threads >> >(d_kasaiMatrix, d_distanceMatrix);
		}
		hipDeviceSynchronize();// ͬ������
		//printD(d_kasaiMatrix, mem_sizeTransportMatrix / 4);
/*
		/// �����Ƿ���ȷ	
		
		h_kasaiMatrix = (float *)malloc(mem_sizeTransportMatrix);
		error = hipMemcpy(h_kasaiMatrix, d_kasaiMatrix, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
			printf("hipMemcpy h_kasaiMatrix<-d_kasaiMatrix, returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			exit(EXIT_FAILURE);
		}
		float *check_kasaiMatrix = (float *)malloc(mem_sizeTransportMatrix);
		printf("kasaiMatrix on GPU and CPU\n ");
		for (int i = 0; i < size_transportMatrix; i++){
			check_kasaiMatrix[i] = exp(-Lamda*c_C[i]);
			printf("%f  %f \n", h_kasaiMatrix[i], check_kasaiMatrix[i]);
		}
		diff_ = 0;
		for (int i = 0; i < size_transportMatrix; i++){
			diff_ += abs(check_kasaiMatrix[i] - h_kasaiMatrix[i]);
		}
		printf("The difference between kasai Matrix of CPU and GPU is %f.\n", diff_);
		//	free(check_kasaiMatrix);
*/		
		// ���㴫��ƻ�����
		/*
		float diff_stopValueU = 9999;
		float stop_valueZero = 9999;
		*/
		int compt = 0;
		int maxIter = 1000;
		float tolerance = 0.001;
		while (compt < maxIter){
			for (int i = 0; i < _iter; i++)
			{
				//  d_kasaiMatrix ��һ��size_original * size_sampling �ľ��� d_V ��һ��size_original������ ***/
				//  ��һ����d_kasaiV ��һ�� size_sampling ������
				stat = hipblasSgemv(handle, HIPBLAS_OP_T, size_originalPoint, size_samplingPoint, &alpha, d_kasaiMatrix, size_originalPoint, d_V, 1, &beta, d_kasaiV, 1);
				if (stat != HIPBLAS_STATUS_SUCCESS){
					printf("hipblasSgemv failed\n");
					exit(EXIT_FAILURE);
				}
				hipDeviceSynchronize();// ͬ������
				//��ʾ����
			//	printD(d_kasaiV, size_samplingPoint);

/*
				// �����ȷ��
	
				float *h_kasaiV = (float *)malloc(mem_sizeSamplingPoint);
				error = hipMemcpy(h_kasaiV, d_kasaiV, mem_sizeSamplingPoint, hipMemcpyDeviceToHost);
				if (error != hipSuccess){
					printf("hipMemcpy (h_kasaiV, d_kasaiV) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
					exit(EXIT_FAILURE);
				}
						
				float *check_kasaiV = (float *)malloc(mem_sizeSamplingPoint);
				printf("kasaiV vector: GPU  CPU\n");
				for (int i = 0; i < size_samplingPoint; i++){
					float temp_ = 0;
					for (int j = 0; j < size_originalPoint; j++){
						temp_ += check_kasaiMatrix[i*size_originalPoint + j] * h_V[j];
					}
					check_kasaiV[i] = temp_;
					printf("KasaiV vector: %f  %f\n", h_kasaiV[i], check_kasaiV[i]);
				}
				diff_ = 0;
				for (int i = 0; i < size_samplingPoint; i++){
				diff_ += abs(h_kasaiV[i] - check_kasaiV[i]);
				}
				printf("The differenc of kasaiV vector: %f\n", diff_);
				//	free(check_kasaiV);
*/
				//�ڶ�����
				threads.x = block_size;
				threads.y = 1;
				threads.z = 1;
				grid.x = ceil(float(size_samplingPoint) / threads.x);
				grid.y = 1;
				grid.z = 1;
				elementWiseDIV<32> << <grid, threads >> >(d_U, d_samplingPointDensity, d_kasaiV);
				hipDeviceSynchronize();// ͬ������
				//����	
				//printD(d_U, size_samplingPoint);
/*
				error = hipMemcpy(h_U, d_U, mem_sizeSamplingPoint, hipMemcpyDeviceToHost);
				for (int i = 0; i < size_samplingPoint; i++){
					printf("CPU_U & h_U vector: %f %f\n", 1.0 / size_samplingPoint / check_kasaiV[i], h_U[i]);
				}
*/	
				//������
				stat = hipblasSgemv(handle, HIPBLAS_OP_N, size_originalPoint, size_samplingPoint, &alpha, d_kasaiMatrix, size_originalPoint, d_U, 1, &beta, d_kasaiU, 1);
				if (stat != HIPBLAS_STATUS_SUCCESS){
					printf("hipblasSdot failed\n");
					exit(EXIT_FAILURE);
				}
				hipDeviceSynchronize();// ͬ������
				//����//
				//printD(d_kasaiU, size_originalPoint);
/*
				error = hipMemcpy(h_kasaiU, d_kasaiU, mem_sizeOriginalPoint, hipMemcpyDeviceToHost);
				if (error != hipSuccess){
					printf("hipMemcpy (h_kasaiV, d_kasaiV) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
					exit(EXIT_FAILURE);
				}  

				for(int i = 0; i < size_originalPoint; i++){
					printf("KasaiU vector: %f \n", h_kasaiU[i]);
				}
*/	
				//���Ĳ�
				threads.x = block_size;
				threads.y = 1;
				threads.z = 1;
				grid.x = ceil(float(size_originalPoint) / threads.x);
				grid.y = 1;
				grid.z = 1;
				elementWiseDIV<32> << <grid, threads >> >(d_V, d_originalPointDensity, d_kasaiU);
				hipDeviceSynchronize();// ͬ������

				compt++;
				//��ʾ����
				//printD(d_V, size_originalPoint);
			}
			//���Ǽ���K'*u����Ϊu�����ˣ�������Ҫ���¼���
			//stat = hipblasSgemv(handle, HIPBLAS_OP_N, size_originalPoint, size_samplingPoint, &alpha, d_kasaiMatrix, size_originalPoint, d_U, 1, &beta, d_kasaiV, 1);
			stat = hipblasSgemv(handle, HIPBLAS_OP_T, size_originalPoint, size_samplingPoint, &alpha, d_kasaiMatrix, size_originalPoint, d_V, 1, &beta, d_kasaiV, 1);
			if (stat != HIPBLAS_STATUS_SUCCESS){
				printf("hipblasSdot failed\n");
				exit(EXIT_FAILURE);
			}
			hipDeviceSynchronize();// ͬ������
			//printD(d_kasaiV, 4);
			//����v.*(K'*u)
			threads.x = block_size;
			threads.y = 1;
			threads.z = 1;
			grid.x = ceil(float(size_samplingPoint) / threads.x);
			grid.y = 1;
			grid.z = 1;
			elementWiseMUL<32> << <grid, threads >> >(d_tempVectorStopCri, d_U, d_kasaiV);
			hipDeviceSynchronize();
			//��ʾ����
			//printD(d_tempVectorStopCri, size_samplingPoint);
			
			//��ȥb
			//stat = hipblasSaxpy(handle, size_originalPoint, &temp_alpha, d_originalPointDensity, 1, d_tempVectorStopCri, 1);
			stat = hipblasSaxpy(handle, size_samplingPoint, &temp_alpha, d_samplingPointDensity, 1, d_tempVectorStopCri, 1);
			if (stat != HIPBLAS_STATUS_SUCCESS){
				printf("hipblasSdot failed\n");
				exit(EXIT_FAILURE);
			}
			hipDeviceSynchronize();// ͬ������
			//��ʾ����
			//printD(d_tempVectorStopCri, size_samplingPoint);

			/// ����u v ��ֵֹͣ,�õ�(v.*(K'*u)-b)�Ķ�����
			//stat = hipblasSnrm2(handle, size_originalPoint, d_tempVectorStopCri, 1, stop_valueU);
			stat = hipblasSnrm2(handle, size_samplingPoint, d_tempVectorStopCri, 1, stop_valueU);
			hipDeviceSynchronize();// ͬ������
		//	printf("stop_valueU: %f \n", *stop_valueU);

			if (*stop_valueU < tolerance || std::isnan(*stop_valueU))
				break;
			/*diff_stopValueU = abs(stop_valueZero - *stop_valueU);
			stop_valueZero = *stop_valueU;
			 diff_stopValueU);*/
	}

/*
		float *h_U = (float *)malloc(mem_sizeSamplingPoint);
		error = hipMemcpy(h_U, d_U, mem_sizeSamplingPoint, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
		printf("hipMemcpy (h_U, d_U) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
		}

		float *h_kasaiU = (float *)malloc(mem_sizeOriginalPoint);
		error = hipMemcpy(h_kasaiU, d_kasaiU, mem_sizeOriginalPoint, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
		printf("hipMemcpy (h_kasaiU, d_kasaiU) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
		}

		error = hipMemcpy(h_V, d_V, mem_sizeOriginalPoint, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
		printf("hipMemcpy (h_V, d_V) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
		}




		float *check_U = (float *)malloc(mem_sizeSamplingPoint);
		printf("temp vector U: GPU    CPU\n");
		for (int i = 0; i < size_samplingPoint; i++){
		check_U[i] = h_samplingPointDensity[i] / h_kasaiV[i];
		printf("%f  %f\n", h_U[i], check_U[i]);
		}

		diff_ = 0;
		for (int i = 0; i < size_samplingPoint; i++){
		diff_ += abs(h_U[i] - check_U[i]);
		}
		printf("the difference of vector U: %f\n", diff_);

		free(check_U);


		float *check_kasaiU = (float *)malloc(mem_sizeOriginalPoint);
		printf("kasaiU vector: GPU  CPU\n");
		for (int i = 0; i < size_originalPoint; i++){
		float temp_ = 0;
		for (int j = 0; j < size_samplingPoint; j++){
		temp_ += h_kasaiMatrix[j*size_originalPoint + i] * h_U[j];
		}
		check_kasaiU[i] = temp_;
		printf("KasaiU vector: %f  %f\n", h_kasaiU[i], check_kasaiU[i]);
		}
		diff_ = 0;
		for (int i = 0; i < size_originalPoint; i++){
		diff_ += abs(h_kasaiU[i] - check_kasaiU[i]);
		}
		printf("The differenc of kasaiU vector: %f\n", diff_);
		free(check_kasaiU);


		float *check_V = (float *)malloc(mem_sizeOriginalPoint);
		printf("temp vector V: GPU    CPU\n");
		for (int i = 0; i < size_originalPoint; i++){
		check_V[i] = h_originalPointDensity[i] / h_kasaiU[i];
		printf("%f  %f\n", h_V[i], check_V[i]);
		}

		diff_ = 0;
		for (int i = 0; i < size_originalPoint; i++){
		diff_ += abs(h_V[i] - check_V[i]);
		}
		printf("the difference of vector V: %f\n", diff_);

		free(check_V);
*/		
		///���㴫��ƻ�����
		// ����hipblasSdgmm�����Ծ���û��op���������������൱��ת�õļ��㣬 A= BCD   AT = DT CT BT (T��ʾת��)�ر�С��
		// �Դ��м������ d_transportMatrix ���󣬸պ���һ��size_originalPoint * size_samplingPoint  �Ҹպ��ǰ��������õľ���
		hipblasSdgmm(handle, HIPBLAS_SIDE_LEFT, size_originalPoint, size_samplingPoint, d_kasaiMatrix, size_originalPoint, d_V, 1, d_diagUKasaiMatrix, size_originalPoint);
		hipDeviceSynchronize();// ͬ������

		hipblasSdgmm(handle, HIPBLAS_SIDE_RIGHT, size_originalPoint, size_samplingPoint, d_diagUKasaiMatrix, size_originalPoint, d_U, 1, d_transportPlan, size_originalPoint);
		hipDeviceSynchronize();// ͬ������
		//����
		//std::cout << "����ƻ�" << std::endl;
		//printD(d_transportPlan, size_transportMatrix);
/*
		// �˶���ȷ��	
		h_transportPlan = (float *)malloc(mem_sizeTransportMatrix);

		error = hipMemcpy(h_transportPlan, d_transportPlan, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
		printf("hipMemcpy (h_transportPlan, d_transportPlan) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
		}
		
		float *check_transportPlan = (float *)malloc(mem_sizeTransportMatrix);
		float *ch_transportPlan = (float *)malloc(mem_sizeTransportMatrix);

		for (int i = 0; i < size_samplingPoint; i++){
		for (int j = 0; j < size_originalPoint; j++){
			ch_transportPlan[i*size_originalPoint + j] = h_U[i] * check_kasaiMatrix[i*size_originalPoint + j];
		}
		}
		for (int i = 0; i < size_samplingPoint; i++){
		for (int j = 0; j < size_originalPoint; j++){
		check_transportPlan[i*size_originalPoint + j] = ch_transportPlan[i*size_originalPoint + j] * h_V[j];
		}
		}

		printf("Transport plan matrix: GPU  CPU \n");
		for (int i = 0; i < size_transportMatrix; i++){
		printf("Transport plan matrix: %f  %f\n", h_transportPlan[i], check_transportPlan[i]);
		}


		//free(check_transportPlan);
		//free(ch_transportPlan);
*/
		/// ��������ֵ����,����Ϊ��������һ����Y��ԽǾ���Ļ��֣��ڶ����Ǽ���ԭʼ��������µľ����������������Ǽ��������ͣ����ֳ�������С��
		//��һ��
		hipblasSdgmm(handle, HIPBLAS_SIDE_RIGHT, size_originalPoint, size_samplingPoint, d_transportPlan, size_originalPoint, d_samplingPointDensity_samplingPoint, 1, d_transportPlanDensity, size_originalPoint);
		hipDeviceSynchronize();// ͬ������
		//�ڶ���
		hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimsA.y, size_samplingPoint, size_originalPoint, &alpha, d_B, dimsA.y, d_transportPlanDensity, size_originalPoint, &beta, d_tempSamplPointCoordinate, dimsA.y);
		hipDeviceSynchronize();// ͬ������
		//��ʾ����
		//std::cout << "�������꣺" << std::endl;
		//printD(d_tempSamplPointCoordinate, 8);
/*
		float *h_transportPlanDensity = (float *)malloc(mem_sizeTransportMatrix);
		error = hipMemcpy(h_transportPlanDensity, d_transportPlanDensity, mem_sizeTransportMatrix, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
		printf("hipMemcpy (h_transportPlanDensity, d_transportPlanDensity) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
		}

		float *h_tempSamplPointCoordinate = (float *)malloc(mem_sizeA);
		error = hipMemcpy(h_tempSamplPointCoordinate, d_tempSamplPointCoordinate, mem_sizeA, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
		printf("hipMemcpy (h_tempSamplPointCoordinate, d_tempSamplPointCoordinate) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
		}
		
		// CPU�˶���ȷ�� h_transportPlanDensity
		float *h_transportPlanDensityT = (float *)malloc(mem_sizeTransportMatrix);
		for (int i = 0; i < size_samplingPoint; i++){
			for (int j = 0; j < size_originalPoint; j++){
				h_transportPlanDensityT[j*size_samplingPoint + i] = h_transportPlanDensity[i*size_originalPoint + j];
			}
		}


		float *check_transportPlanDensity = (float *)malloc(mem_sizeTransportMatrix);
		float *check_transportPlanDensityT = (float *)malloc(mem_sizeTransportMatrix);
		float *check_tempSamplPointCoordinate = (float *)malloc(mem_sizeA);
		float *check_transportPlanT = (float *)malloc(mem_sizeTransportMatrix);
		for (int i = 0; i < size_samplingPoint; i++){
			for (int j = 0; j < size_originalPoint; j++){
				check_transportPlanT[j*size_samplingPoint + i] = check_transportPlan[i*size_originalPoint + j];
			}
		}

		for (int i = 0; i < size_originalPoint; i++){
			for (int j = 0; j < size_samplingPoint; j++){
				check_transportPlanDensity[i*size_samplingPoint + j] = check_transportPlanT[i*size_samplingPoint + j] / h_samplingPointDensity[j];
			}
		}

		for (int i = 0; i < size_originalPoint; i++){
			for (int j = 0; j < size_samplingPoint; j++){
				check_transportPlanDensityT[j* size_originalPoint + i] = check_transportPlanDensity[i*size_samplingPoint + j];
			}
		}

		printf("Tempt matrix h_transportPlanDensity: GPU   CPU\n");
		for (int i = 0; i < size_transportMatrix; i++){
		printf("Tempt matrix h_transportPlanDensity:  %f  %f\n", h_transportPlanDensity[i], check_transportPlanDensityT[i]);
		}
*/
/*		// �˶���ȷ�� ����
	
		float *h_BT = (float *)malloc(mem_sizeB);
		float *check_ordinate = (float*)malloc(mem_sizeA);
		for (int i = 0; i < size_originalPoint; i++){
		for (int j = 0; j < dimsB.y; j++){
		h_BT[j*size_originalPoint + i] = h_B[i*dimsB.y + j];
		}
		}


		for (int i = 0; i < dimsB.y; i++){
		for (int j = 0; j < size_samplingPoint; j++){
			check_ordinate[i * size_samplingPoint + j] = 0;
		for (int k = 0; k < size_originalPoint; k++){
		check_ordinate[i * size_samplingPoint + j] += h_BT[i * size_originalPoint + k] * check_transportPlanDensity[k * size_samplingPoint + j];
		}
		}
		}



		float *h_tempSamplPointCoordinateT = (float *)malloc(mem_sizeA);
		for (int i = 0; i < dimsA.y; i++){
		for (int j = 0; j < size_samplingPoint; j++){
		h_tempSamplPointCoordinateT[i * size_samplingPoint + j] = h_tempSamplPointCoordinate[j * dimsA.y + i];
		}
		}

		printf("Cordinate Y x Kasai x diag��gi�� on GPU  CPU\n");
		for (int i = 0; i < dimsA.y * size_samplingPoint; i++){
		printf("Coordinate: %f  %f \n", h_tempSamplPointCoordinateT[i], check_ordinate[i]);
		}
*/
		//�����������µĲ���������
		hipblasSaxpy(handle, size_samplingPoint * dimsA.y, minusOne_divLabR, d_A, 1, d_tempSamplPointCoordinate, 1);
		hipDeviceSynchronize();// ͬ������

		hipblasSaxpy(handle, size_samplingPoint * dimsA.y, theta_lambR, d_tempSamplPointCoordinate, 1, d_A, 1);
		hipDeviceSynchronize();// ͬ������
		//��ʾ����
		//std::cout << "�����ɵ��������꣺" << std::endl;
		//printD(d_A, size_A);

/*
		float *h_samplPointCoordinate = (float *)malloc(mem_sizeA);
		error = hipMemcpy(h_samplPointCoordinate, d_A, mem_sizeA, hipMemcpyDeviceToHost);
		if (error != hipSuccess){
		printf("hipMemcpy (h_tempSamplPointCoordinate, d_tempSamplPointCoordinate) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
		}

		float *h_AT = (float *)malloc(mem_sizeA);
		for (int i = 0; i < size_samplingPoint; i++){
			for (int j = 0; j < dimsA.y; j++){
				h_AT[j*size_samplingPoint + i] = h_A[i*dimsA.y + j];
			}
		}

		for (int i = 0; i < size_samplingPoint * dimsA.y; i++){
			float temp_ = check_ordinate[i];
			check_ordinate[i] = *one_minusTheta * h_AT[i] + *theta_lambR * temp_;
		}

		float *h_samplPointCoordinateT = (float *)malloc(mem_sizeA);
		for (int i = 0; i < size_samplingPoint; i++){
			for (int j = 0; j < dimsA.y; j++){
				h_samplPointCoordinateT[j * size_samplingPoint + i] = h_samplPointCoordinate[i * dimsA.y + j];
			}
		}

		printf("the updated coordinate: GPU  CPU \n");
		for (int i = 0; i < size_samplingPoint*dimsA.y; i++){
			printf("the updated coordinate: %f  %f \n", h_samplPointCoordinateT[i], check_ordinate[i]);
		}
*/		
		// ���¾������
		//block_size = 4;
		//	dim3 threads(1, block_size, 1);
		//		dim3 grid(dimsA.x, dimsB.x / block_size, 1);

		threads.x = 1;
		threads.y = block_size;
		threads.z = 1;
		grid.x = dimsA.x;
		grid.y = dimsB.x / block_size;
		grid.z = 1;

		// ����������
		if (block_size == 16){
			distancePointToPointCUDA<4> << <grid, threads >> >(d_distanceMatrix, d_A, d_B, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
		}
		else{
			distancePointToPointCUDA<32> << <grid, threads >> >(d_distanceMatrix, d_A, d_B, dimsA.x, dimsA.y, dimsB.x, dimsB.y);
		}
		hipDeviceSynchronize();// ͬ������

		//// ���㴫����۵�
		
		hipblasSdot(handle, size_transportMatrix, d_distanceMatrix, 1, d_transportPlan, 1, transportCost);
		hipDeviceSynchronize();// ͬ������
		//printf("������ۣ� %f\n", *transportCost);

		*stop_valueX =*preTransportCost - *transportCost;
		*preTransportCost = *transportCost;
		if (stop_valueX <= 0)
			break;
		
}

	//ͳ��ʱ��
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	

	/// ������ĵ�
	float *h_samplPointCoordinate = (float *)malloc(mem_sizeA);
	error = hipMemcpy(h_samplPointCoordinate, d_A, mem_sizeA, hipMemcpyDeviceToHost);
	if (error != hipSuccess){
		printf("hipMemcpy (h_tempSamplPointCoordinate, d_tempSamplPointCoordinate) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	//�����������
	printH(h_samplPointCoordinate, size_A);
	writeResult(d_A, size_A,0);
	// CUBLAS handle

	stat = hipblasDestroy(handle);
	if (stat != HIPBLAS_STATUS_SUCCESS){
		printf("hipblasDestroy failed\n");
		exit(EXIT_FAILURE);
	}
	std::cout << "����ʱ��Ϊ��" << time / CLOCKS_PER_SEC << std::endl;
	// Record the stop event
	/*error = hipEventRecord(stop, NULL);
	if (error != hipSuccess){
		fprintf(stderr, " Failed to record stop event ( error code %s)! \n", hipGetErrorString(error));
	}

	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)! \n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);

	if (error != hipSuccess){
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
	*/
	/*	for (int i = 0; i < dimsA.x; i++){
	for (int j = 0; j < dimsA.y; j++){
	printf("%f  ", h_A[j*dimsA.x + i]);
	}
	printf("\n");
	}
	*/
	/*	for (int i = 0; i < dimsA.x * dimsA.y; i++){

	printf("%f  ", h_A[ i]);
	}
	printf("\n");


	for (int i = 0; i < dimsB.x; i++){
	for (int j = 0; j < dimsB.y; j++){
	printf("%f  ", h_B[ i * dimsB.y + j]);
	}
	printf("\n");
	}

	for (int i = 0; i < dimsA.x; i++){
	for (int j = 0; j < dimsB.x; j++){
	//		printf("%f  ", h_C[i*dimsB.x + j]);
	}
	printf("\n");
	}

	for (int i = 0; i < dimsB.x; i++){
	for (int j = 0; j < dimsB.y; j++){
	//			printf("%f  ", h_D[i * dimsB.y + j]);
	}
	printf("\n");
	}
	*/
	// Check  the result
	/*
	float *c_C = (float *)malloc(mem_sizeC);
	for (int i = 0; i < dimsA.x; i++){
	for (int j = 0; j < dimsB.x; j++){
	float diff_x;
	float diff_y;
	diff_x = *(h_A + i * dimsA.y) - *(h_B + j*dimsB.y );
	diff_y = *(h_A + i * dimsA.y + 1) - *(h_B + j*dimsB.y + 1);
	*(c_C + dimsB.x * i + j) = diff_x * diff_x + diff_y * diff_y;
	}
	}

	float diff_ = 0;
	for (int i = 0; i < dimsA.x; i++){
	for (int j = 0; j < dimsB.x; j++)
	diff_ += ( h_C[i * dimsB.x + j] - c_C[i*dimsB.x + j]) * (h_C[i * dimsB.x + j] - c_C[i*dimsB.x + j] );

	}
	*/

	//printf("The difference between results of CPU and GPU is %f.\n", diff_);

	//printf("The dot product of h_B is %f\n", result_);

	//Clean up memory
	//free(h_A);
	//free(h_B);
	//free(h_C);
	//free(h_D);
	//free(d_transportPlan);
	hipFree(d_A);
	hipFree(d_B);
	//hipFree(d_C);

	printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits

	hipDeviceReset();

	return EXIT_SUCCESS;

}


// set variables for distance compuation between points
void setVariableForDistanceComputation(){




	return;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	int *dev_d = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_d, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);



	addKernel << <1, size >> >(dev_d, dev_c, dev_b);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_d, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}

/*
hipError_t multiMatriWithCuda(float *c, float *a, float *b, int widthA, int heightA, int widthB, int heightB)
{
float2 *dev_a = 0;
float2 *dev_b = 0;
float2 *dev_c = 0;

hipError_t cudaStatus;

// Choose which GPU to run on, change this on a multi-GPU system.
cudaStatus = hipSetDevice(0);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
goto Error;
}

// Allocate GPU buffers for three vectors (two input, one output)    .
cudaStatus = hipMalloc((void**)&dev_c, heightA * heightB * sizeof(int));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed!");
goto Error;
}

cudaStatus = hipMalloc((void**)&dev_a, heightA * widthA * sizeof(int));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed!");
goto Error;
}

cudaStatus = hipMalloc((void**)&dev_b, heightB * widthB * sizeof(int));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed!");
goto Error;
}



// Copy input vectors from host memory to GPU buffers.
cudaStatus = hipMemcpy(dev_a, a, widthA * heightA * sizeof(int), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy failed!");
goto Error;
}

cudaStatus = hipMemcpy(dev_b, b, widthB * heightB * sizeof(int), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy failed!");
goto Error;
}

// Launch a kernel on the GPU with one thread for each element.


// Check for any errors launching the kernel
cudaStatus = hipGetLastError();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
goto Error;
}

// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
goto Error;
}

// Copy output vector from GPU buffer to host memory.
cudaStatus = hipMemcpy(c, dev_d, size * sizeof(int), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy failed!");
goto Error;
}

Error:
hipFree(dev_c);
hipFree(dev_a);
hipFree(dev_b);

return cudaStatus;
}
*/